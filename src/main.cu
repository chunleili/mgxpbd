#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <sstream>
#include <cstdio>
#include <cmath>
#include <chrono>
#include <filesystem>
#include <vector>

#include <hip/hip_runtime.h>
#include <thrust/universal_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "helper_timer.h"

#include <Eigen/Dense>
#include <Eigen/Sparse>
#include <unsupported/Eigen/SparseExtra>
#include <igl/readOBJ.h>
#include <igl/writeOBJ.h>
#include <glm/glm.hpp>

using namespace std;

// constants
const int N = 256;
const int NV = (N + 1) * (N + 1);
const int NT = 2 * N * N;
const int NE = 2 * N * (N + 1) + N * N;
const float h = 0.01;
const int M = NE;
const int new_M = int(NE / 100);
const double compliance = 1.0e-8;
const double alpha = compliance * (1.0 / h / h);
const float omega = 0.5; //under-relaxing factor

// control variables
std::string proj_dir_path;
unsigned num_particles = 0;
unsigned frame_num = 0;
unsigned end_frame = 1000;
unsigned max_iter = 50;
std::string out_dir = "./result/cloth3d_256_50_amg/";
bool output_mesh = true;

// typedefs
using Vec3f = glm::vec3;
using Vec2i = glm::ivec2;
using Vec3i = glm::ivec3;
using Field1f = vector<float>;
using Field3f = vector<Vec3f>;
using Field3i = vector<Vec3i>;
using Field2i = vector<Vec2i>;
using Field1i = vector<int>;

// global fields
Field3f pos;
Field2i edge;
Field1i tri;
Field1f rest_len;
Field3f vel;
Field1f inv_mass;
Field1f lagrangian;
Field1f constraints;
Field1f dLambda;
Field3f pos_mid;
Field3f acc_pos;
Field3f old_pos;

// we have to use pos_vis for visualization because libigl uses Eigen::MatrixXd
Eigen::MatrixXd pos_vis;
Eigen::MatrixXi tri_vis;

Eigen::SparseMatrix<float> R, P;
Eigen::SparseMatrix<float> M_inv(3 * NV, 3 * NV);
Eigen::SparseMatrix<float> ALPHA(M,M);

// utility functions
__forceinline float length(Vec3f& vec)
{
    return glm::length(vec);
}

__forceinline Vec3f normalize(Vec3f& vec)
{
    return glm::normalize(vec);
}

std::string get_proj_dir_path()
{
    std::filesystem::path p(__FILE__);
    std::filesystem::path prj_path = p.parent_path().parent_path();
    proj_dir_path = prj_path.string();

    std::cout << "Project directory path: " << proj_dir_path << std::endl;
    return proj_dir_path;
}
// this code run before main, in case of user forget to call get_proj_dir_path()
static string proj_dir_path_pre_get = get_proj_dir_path();

/** @brief A parallel for loop. It should be used with a lambda function.
 * Learn from https://github.com/parallel101/course/blob/2d30da61b442008c003f69225e6feca20a4ca7df/08/06_thrust/01/main.cu
 * Usage:
 * // add one to each vertex
 * parallel_for<<<num_particles / 512, 128>>>(num_particles, [pos = pos.data()] __device__ (int i) {
 *    pos[i].y += 1.0;
 * });
 * checkCudaErrors(hipDeviceSynchronize());
 *
 */
template <typename Func>
__global__ void parallel_for(int n, Func func)
{
    for (int i = blockDim.x * blockIdx.x + threadIdx.x;
         i < n; i += blockDim.x * gridDim.x)
    {
        func(i);
    }
}

/// @brief Usage: Timer t("timer_name");
///               t.start();
///               //do something
///               t.end();
class Timer
{
private:
    std::chrono::time_point<std::chrono::steady_clock> m_start;
    std::chrono::time_point<std::chrono::steady_clock> m_end;

public:
    std::string name = "";
    Timer(std::string name = "") : name(name){};
    inline void start()
    {
        m_start = std::chrono::steady_clock::now();
    };
    inline void end(string msg = "", string unit="ms")
    {
        m_end = std::chrono::steady_clock::now();
        if(unit == "ms")
        {
            std::chrono::duration<double, std::milli> elapsed = m_end - m_start;
            printf("%s(%s timer): %.0f(ms)\n", msg.c_str(), name.c_str(), elapsed.count());
        }
        else if(unit == "s")
        {
            std::chrono::duration<double> elapsed = m_end - m_start;
            printf("%s(%s timer): %.0f(s)\n", msg.c_str(), name.c_str(), elapsed.count());
        }
    }
    inline void reset()
    {
        m_start = std::chrono::steady_clock::now();
        m_end = std::chrono::steady_clock::now();
    };
};
Timer global_timer("global");
Timer t_sim("sim"), t_main("main"), t_substep("substep"), t_init("init");

/// @brief Usage: SdkTimer t("timer_name");
///               t.start();
///               //do something
///               t.end();
class SdkTimer
{
private:
    StopWatchInterface *m_timer = NULL;

public:
    std::string name = "";
    SdkTimer(std::string name_ = "") : name(name_)
    {
        sdkCreateTimer(&m_timer);
    }
    SdkTimer::~SdkTimer()
    {
        sdkDeleteTimer(&m_timer);
    }

    inline void start()
    {
        sdkStartTimer(&m_timer);
    }

    inline void end()
    {
        sdkStopTimer(&m_timer);
        printf("%s time elapsed: %.4f(ms)\n", name.c_str(), sdkGetTimerValue(&m_timer));
        sdkResetTimer(&m_timer);
    };

    inline void reset()
    {
        sdkResetTimer(&m_timer);
    };
};

// caution: the tic toc cannot be nested
inline void tic()
{
    global_timer.reset();
    global_timer.start();
}

inline void toc(string message = "")
{
    global_timer.end(message);
    global_timer.reset();
}

void copy_pos_to_pos_vis()
{
    // copy pos to pos_vis
    for (int i = 0; i < num_particles; i++)
    {
        pos_vis(i, 0) = pos[i][0];
        pos_vis(i, 1) = pos[i][1];
        pos_vis(i, 2) = pos[i][2];
    }
}

/* -------------------------------------------------------------------------- */
/*                            simulation functions                            */
/* -------------------------------------------------------------------------- */
void init_edge()
{
    for (int i = 0; i < N + 1; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int edge_idx = i * N + j;
            int pos_idx = i * (N + 1) + j;
            edge[edge_idx][0] = pos_idx;
            edge[edge_idx][1] = pos_idx + 1;
        }
    }

    int start = N * (N + 1);
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N + 1; j++)
        {
            int edge_idx = start + j * N + i;
            int pos_idx = i * (N + 1) + j;
            edge[edge_idx][0] = pos_idx;
            edge[edge_idx][1] = pos_idx + N + 1;
        }
    }

    start = 2 * N * (N + 1);
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int edge_idx = start + i * N + j;
            int pos_idx = i * (N + 1) + j;
            if ((i + j) % 2 == 0)
            {
                edge[edge_idx][0] = pos_idx;
                edge[edge_idx][1] = pos_idx + N + 2;
            }
            else
            {
                edge[edge_idx][0] = pos_idx + 1;
                edge[edge_idx][1] = pos_idx + N + 1;
            }
        }
    }

    for (int i = 0; i < NE; i++)
    {
        int idx1 = edge[i][0];
        int idx2 = edge[i][1];
        Vec3f p1 = pos[idx1];
        Vec3f p2 = pos[idx2];
        rest_len[i] = length(p1 - p2);
    }
}

void semi_euler()
{
    Vec3f gravity = Vec3f(0.0, -0.1, 0.0);
    for (int i = 0; i < num_particles; i++)
    {
        if (inv_mass[i] != 0.0)
        {
            vel[i] += h * gravity;
            old_pos[i] = pos[i];
            pos[i] += h * vel[i];
        }
    }
}

void reset_lagrangian()
{
    for (int i = 0; i < NE; i++)
    {
        lagrangian[i] = 0.0;
    }
}

void reset_accpos()
{
    for (int i = 0; i < num_particles; i++)
    {
        acc_pos[i] = Vec3f(0.0, 0.0, 0.0);
    }
}

void solve_constraints_xpbd()
{
    for (int i = 0; i < NE; i++)
    {
        int idx0 = edge[i][0];
        int idx1 = edge[i][1];
        float invM0 = inv_mass[idx0];
        float invM1 = inv_mass[idx1];
        Vec3f dis = pos[idx0] - pos[idx1];
        float constraint = length(dis) - rest_len[i];
        Vec3f gradient = normalize(dis);
        float l = -constraint / (invM0 + invM1);
        float delta_lagrangian = -(constraint + lagrangian[i] * alpha) / (invM0 + invM1 + alpha);
        lagrangian[i] += delta_lagrangian;
        if (invM0 != 0.0)
        {
            acc_pos[idx0] += invM0 * delta_lagrangian * gradient;
        }
        if (invM1 != 0.0)
        {
            acc_pos[idx1] -= invM1 * delta_lagrangian * gradient;
        }
    }
}

void update_pos()
{
    for (int i = 0; i < num_particles; i++)
    {
        if (inv_mass[i] != 0.0)
        {
            pos[i] += omega * acc_pos[i];
        }
    }
}

void collision()
{
    for (int i = 0; i < num_particles; i++)
    {
        if (pos[i].z < -2.0)
        {
            pos[i].z = 0.0;
        }
    }
}

void update_vel()
{
    for (int i = 0; i < num_particles; i++)
    {
        if (inv_mass[i] != 0.0)
        {
            vel[i] = (pos[i] - old_pos[i]) / h;
        }
    }
}

void substep_xpbd()
{
    semi_euler();
    reset_lagrangian();
    for (int i = 0; i <= max_iter; i++)
    {
        // printf("iter = %d\n", i);
        reset_accpos();
        solve_constraints_xpbd();
        update_pos();
        collision();
    }
    update_vel();
}



void fill_M_inv()
{
    typedef Eigen::Triplet<float> T;

    std::vector<T> inv_mass_3(3*NV);
    for(int i=0; i < 3*NV; i++)
    {
        inv_mass_3[i] = T(i, i, inv_mass[int(i/3)]);
    }
    M_inv.setFromTriplets(inv_mass_3.begin(), inv_mass_3.end());
}

void fill_ALPHA()
{
    typedef Eigen::Triplet<float> T;

    std::vector<T> alpha_(NE);
    for(int i=0; i < NE; i++)
    {
        alpha_[i] = T(i, i, alpha);
    }
    ALPHA.setFromTriplets(alpha_.begin(), alpha_.end());
}

void substep_all_solver()
{
    semi_euler();
    reset_lagrangian();

}


void main_loop()
{
    for (frame_num = 0; frame_num <= end_frame; frame_num++)
    {
        printf("---------\n");
        printf("frame_num = %d\n", frame_num);

        t_substep.start();
        // substep_xpbd();
        substep_all_solver();
        t_substep.end();

        if (output_mesh)
        {
            tic();
            std::string out_mesh_name = proj_dir_path + "/results/" + std::to_string(frame_num) + ".obj";

            printf("output mesh: %s\n", out_mesh_name.c_str());
            copy_pos_to_pos_vis();
            igl::writeOBJ(out_mesh_name, pos_vis, tri_vis);
            toc("output mesh");
        }

        printf("frame_num = %d done\n", frame_num);
        printf("---------\n\n");
    }
}

void load_R_P()
{
    // load R, P
    Eigen::loadMarket(R, proj_dir_path + "/data/misc/R.mtx");
    Eigen::loadMarket(P, proj_dir_path + "/data/misc/P.mtx");

    std::cout << "R: " << R.rows() << " " << R.cols() << std::endl;
    std::cout << "P: " << P.rows() << " " << P.cols() << std::endl;
}

void resize_fields()
{
    pos.resize(num_particles);
    edge.resize(NE);
    rest_len.resize(NE);
    vel.resize(num_particles);
    inv_mass.resize(num_particles);
    lagrangian.resize(NE);
    constraints.resize(NE);
    dLambda.resize(NE);
    pos_mid.resize(num_particles);
    acc_pos.resize(num_particles);
    old_pos.resize(num_particles);
    tri.resize(3 * NT);

    tri_vis.resize(NT, 3);
    pos_vis.resize(num_particles, 3);
}

void init_pos()
{
    for (int i = 0; i < N + 1; i++)
    {
        for (int j = 0; j < N + 1; j++)
        {
            int idx = i * (N + 1) + j;
            // pos[idx] = ti.Vector([i / N,  j / N, 0.5])  # vertical hang
            pos[idx] = Vec3f(i / float(N), 0.5, j / float(N)); // horizontal hang
            inv_mass[idx] = 1.0;
        }
    }
    inv_mass[N] = 0.0;
    inv_mass[NV - 1] = 0.0;
}

void init_tri()
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; ++j)
        {
            int tri_idx = 6 * (i * N + j);
            int pos_idx = i * (N + 1) + j;
            if ((i + j) % 2 == 0)
            {
                tri[tri_idx + 0] = pos_idx;
                tri[tri_idx + 1] = pos_idx + N + 2;
                tri[tri_idx + 2] = pos_idx + 1;
                tri[tri_idx + 3] = pos_idx;
                tri[tri_idx + 4] = pos_idx + N + 1;
                tri[tri_idx + 5] = pos_idx + N + 2;
            }
            else
            {
                tri[tri_idx + 0] = pos_idx;
                tri[tri_idx + 1] = pos_idx + N + 1;
                tri[tri_idx + 2] = pos_idx + 1;
                tri[tri_idx + 3] = pos_idx + 1;
                tri[tri_idx + 4] = pos_idx + N + 1;
                tri[tri_idx + 5] = pos_idx + N + 2;
            }
        }
    }

    // reshape tri from 3*NT to (NT, 3)
    for (int i = 0; i < NT; i++)
    {
        int tri_idx = 3 * i;
        int pos_idx = 3 * i;
        tri_vis(i, 0) = tri[tri_idx + 0];
        tri_vis(i, 1) = tri[tri_idx + 1];
        tri_vis(i, 2) = tri[tri_idx + 2];
    }
}

void test()
{
    Eigen::saveMarket(M_inv, "M.mtx");
    Eigen::saveMarket(ALPHA, "ALPHA.mtx");
    Eigen::saveMarket(R, "RR.mtx");
    Eigen::saveMarket(P, "PP.mtx");
}

void run_simulation()
{
    printf("run_simulation\n");

    t_sim.start();

    t_init.start();
    resize_fields();
    init_pos();
    init_edge();
    init_tri();
    load_R_P();
    fill_M_inv();
    fill_ALPHA();
    // test();
    t_init.end();

    main_loop();

    t_sim.end();
}

int main(int argc, char *argv[])
{
    t_main.start();

    // igl::readOBJ(proj_dir_path + "/data/models/cloth.obj", pos_vis, tri);
    // num_particles = pos_vis.rows();
    num_particles = NV;
    printf("num_particles = %d\n", num_particles);

    run_simulation();

    copy_pos_to_pos_vis();

    // igl::writeOBJ(proj_dir_path + "/data/models/bunny2.obj", pos_vis, tri);

    t_main.end("","s");
}